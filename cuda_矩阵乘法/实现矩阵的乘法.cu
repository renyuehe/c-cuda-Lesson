#include "hip/hip_runtime.h"

typedef struct {
    int width;
	int height;
    int pitch;
    float* elements;
} Matrix;


// Step 1: Matrix Data Transfers
// Allocate the device memory where we will copy M to
Matrix Md;
Md.width  = WIDTH;
Md.height = WIDTH;
Md.pitch  = WIDTH;
int size = WIDTH * WIDTH * sizeof(float);
hipMalloc((void**)&Md.elements, size);

// Copy M from the host to the device
// hipMemcpy(destination,source,size,cudaMemcpySourceToDestination);
hipMemcpy(Md.elements, M.elements, size, hipMemcpyHostToDevice);


// Read M from the device to the host into P
hipMemcpy(P.elements, Md.elements, size, hipMemcpyDeviceToHost);
...
// Free device memory
hipFree(Md.elements);




// Matrix multiplication on the (CPU) host in double precision
// for simplicity, we will assume that all dimensions are equal

void MatrixMulOnHost(const Matrix M, const Matrix N, Matrix P)
{   
    for (int i = 0; i < M.height; ++i)
        for (int j = 0; j < N.width; ++j) {
            double sum = 0;
            for (int k = 0; k < M.width; ++k) {
                double a = M.elements[i * M.width + k];
                double b = N.elements[k * N.width + j];
                sum += a * b;
            }
            P.elements[i * N.width + j] = sum;
        }
}






int main(void) {
// Allocate and initialize the matrices
    Matrix  M  = AllocateMatrix(WIDTH, WIDTH, 1);
    Matrix  N  = AllocateMatrix(WIDTH, WIDTH, 1);
    Matrix  P  = AllocateMatrix(WIDTH, WIDTH, 0);

// M * N on the device
    MatrixMulOnDevice(M, N, P);

// Free matrices
    FreeMatrix(M);
    FreeMatrix(N);
    FreeMatrix(P);
return 0;
}






// Matrix multiplication on the device
void MatrixMulOnDevice(const Matrix M, const Matrix N, Matrix P)
{
    // Load M and N to the device
    Matrix Md = AllocateDeviceMatrix(M);
    CopyToDeviceMatrix(Md, M);
    Matrix Nd = AllocateDeviceMatrix(N);
    CopyToDeviceMatrix(Nd, N);

    // Allocate P on the device
    Matrix Pd = AllocateDeviceMatrix(P);
    CopyToDeviceMatrix(Pd, P); // Clear memory

   
	// Setup the execution configuration
    dim3 dimBlock(WIDTH, WIDTH);
    dim3 dimGrid(1, 1);

/*****************************************************************************************/
	//�˴�����Ϊ:host call device ���к���MatrixMulKernel
/*****************************************************************************************/
    // Launch the device computation threads!
    MatrixMulKernel<<<dimGrid, dimBlock>>>(Md, Nd, Pd);

    // Read P from the device
    CopyFromDeviceMatrix(P, Pd); 

    // Free device matrices
    FreeDeviceMatrix(Md);
    FreeDeviceMatrix(Nd);
    FreeDeviceMatrix(Pd);
}





/*****************************************************************************************/
/���Ĵ��벿��,ʵ���˾���ĳ˷�����/���Ĵ��벿��,ʵ���˾���ĳ˷�����
/*****************************************************************************************/
/
// Matrix multiplication kernel �Cthread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{
    // 2D Thread ID
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Pvalue is used to store the element of the matrix
    // that is computed by the thread
    float Pvalue = 0;


	// ���м��������ڴ�,���thread���н���
	// ÿ��thread����һ����һ�еĳ˻�	
    for (int k = 0; k < M.width; ++k)
    { 
         float Melement = M.elements[ty * M.pitch + k];
         float Nelement = Nd.elements[k * N.pitch + tx];
         Pvalue += Melement * Nelement;
    }
    // Write the matrix to device memory;
    // each thread writes one element
    P.elements[ty * P.pitch + tx] = Pvalue;
}

/*****************************************************************************************/

/*****************************************************************************************/






// Allocate a device matrix of same size as M.
Matrix AllocateDeviceMatrix(const Matrix M)
{
    Matrix Mdevice = M;
    int size = M.width * M.height * sizeof(float);
    hipMalloc((void**)&Mdevice.elements, size);
    return Mdevice;
}

// Free a device matrix.
void FreeDeviceMatrix(Matrix M) {
    hipFree(M.elements);
}

void FreeMatrix(Matrix M) {
    free(M.elements);
}



// Copy a host matrix to a device matrix.
void CopyToDeviceMatrix(Matrix Mdevice, const Matrix Mhost)
{
    int size = Mhost.width * Mhost.height * sizeof(float);
    hipMemcpy(Mdevice.elements, Mhost.elements, size, 
	hipMemcpyHostToDevice);
}

// Copy a device matrix to a host matrix.
void CopyFromDeviceMatrix(Matrix Mhost, const Matrix Mdevice)
{
    int size = Mdevice.width * Mdevice.height * sizeof(float);
    hipMemcpy(Mhost.elements, Mdevice.elements, size, 
	hipMemcpyDeviceToHost);
}
