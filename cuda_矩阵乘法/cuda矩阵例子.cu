#include "hip/hip_runtime.h"

/*
����CUDA����Ⱥ��meteor�����˸��������� 
����
x  x  x  x  y  y  y  y 
x  x  x  x  y  y  y  y 
x  x  x  x  y  y  y  y 
x  x  x  x  y  y  y  y 
z  z  z  z  a  a  a  a 
z  z  z  z  a  a  a  a 
z  z  z  z  a  a  a  a 
z  z  z  z  a  a  a  a 

����������� 
�����x��Щ����һ,��y��Щ���Ӷ� 
��z��Щ�����,��a��Щ������ 

����meteor�������֣����Ż���ѧϰ����������д��һ�κܼܺ򵥵ĳ����������������ϣ����meteor���������� 
*/
[quote] 

C/C++ code
#include <stdio.h>

__global__ void testkernel(int *d_A, size_t size)
{
    int dx = blockDim.x * blockIdx.x + threadIdx.x;
    int dy = blockDim.y * blockIdx.y + threadIdx.y;

    if( blockIdx.x == 0 && blockIdx.y == 0 )
       d_A[dx*size+dy] += 1;
    if( blockIdx.x == 0 && blockIdx.y == 1 )
       d_A[dx*size+dy] += 2;
    if( blockIdx.x == 1 && blockIdx.y == 0 )
       d_A[dx*size+dy] += 3;
    if( blockIdx.x == 1 && blockIdx.y == 1 )
       d_A[dx*size+dy] += 4;
}

int main( int argc, char** argv) 
{
int h_A[8][8] = {{1,1,1,1,2,2,2,2},
                 {1,1,1,1,2,2,2,2},
                 {1,1,1,1,2,2,2,2},
                 {1,1,1,1,2,2,2,2},
                 {3,3,3,3,4,4,4,4},
                 {3,3,3,3,4,4,4,4},
                 {3,3,3,3,4,4,4,4},
                 {3,3,3,3,4,4,4,4}};
 
int  *d_A, *h_B;
size_t size = 8 * 8 * sizeof(int);
size_t rsize = 8;
dim3 dimgrid(2,2);
dim3 dimblock(4,4);

h_B = (int*)malloc(size);

hipMalloc( (void **) &d_A, size );
hipMemcpy( d_A, h_A, size, hipMemcpyHostToDevice );

testkernel<<<dimgrid,dimblock>>>(d_A,rsize);

hipMemcpy( h_B, d_A, size, hipMemcpyDeviceToHost );

for(int i = 0; i < 8; i++)
{
  for(int j = 0;j < 8; j++)
      printf("%2d ",h_B[i*rsize+j]);
printf("\n");
}

hipFree(d_A);
free(h_B);
}



[/quote] 

����meteor�ֲ����blockDim.x��threadIdx.x�����������������ӽ��ͣ��������μ�Programme Guide 
blockDim����ָblock��ά��,����ÿ��block��4*4��,����blockDim.x=4 blockDim.y = 4 
threadIdx����ָblock����̵߳�������,����ÿblock��4*4ά��,ÿ��block����16��thread,ÿ��thread��threadIdx.x��0��3,threadIdx.y��0��3,������һ��,������������? 

���ϳ������ͨ����������